#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include <hip/hip_runtime.h>

#define NUM_RUNS 10

#define CUDA_CHECK(func)                                                     	   \
    do {                                                                           \
        hipError_t status = (func);                                               \
        if (status != hipSuccess) {                                               \
            printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__,   \
                hipGetErrorString(status), status);                               \
            exit(EXIT_FAILURE);                                                    \
        }                                                                          \
    } while (0)

#define CHECK(name) \
    float *d_Aref_ ## name, *d_Bref_ ## name, *d_Cref_ ## name; \
    std::cerr << "checking " << #name << std::endl; \
    CUDA_CHECK(hipMalloc(&d_Aref_ ## name, Ref::M * Ref::K * sizeof(float))); \
    CUDA_CHECK(hipMalloc(&d_Bref_ ## name, Ref::K * Ref::N * sizeof(float))); \
    CUDA_CHECK(hipMalloc(&d_Cref_ ## name, Ref::M * Ref::N * sizeof(float))); \
    CUDA_CHECK(hipMemcpy(d_Aref_ ## name, ref.A, Ref::M * Ref::K * sizeof(float), hipMemcpyHostToDevice)); \
    CUDA_CHECK(hipMemcpy(d_Bref_ ## name, ref.B, Ref::K * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
    float* d_Cref_INI_ ## name = new float[M * N](); \
    for (int i = 0; i < Ref::M; i++) { \
        for (int j = 0; j < Ref::N; j++) { \
            d_Cref_INI_ ## name[i * Ref::N + j] = 0; \
        } \
    } \
    CUDA_CHECK(hipMemcpy(d_Cref_ ## name, d_Cref_INI_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
    name(d_Aref_ ## name, d_Bref_ ## name, d_Cref_ ## name, Ref::M, Ref::N, Ref::K); \
    hipError_t err_c_ ## name = hipGetLastError(); \
    if (err_c_ ## name != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err_c_ ## name) << std::endl; \
    } \
    CUDA_CHECK(hipMemcpy(refC, d_Cref_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyDeviceToHost)); \
    if (!ref.checkRef(refC)){ \
        std::cerr << "check ref failed!" << std::endl; \
    };

#define TIME(name) \
    float *d_A_ ## name, *d_B_ ## name, *d_C_ ## name; \
    CUDA_CHECK(hipMalloc(&d_A_ ## name, M * K * sizeof(float))); \
    CUDA_CHECK(hipMalloc(&d_B_ ## name, K * N * sizeof(float))); \
    CUDA_CHECK(hipMalloc(&d_C_ ## name, M * N * sizeof(float))); \
    CUDA_CHECK(hipMemcpy(d_A_ ## name, A, M * K * sizeof(float), hipMemcpyHostToDevice)); \
    CUDA_CHECK(hipMemcpy(d_B_ ## name, B, K * N * sizeof(float), hipMemcpyHostToDevice)); \
    hipEvent_t start_ ## name, end_ ## name; \
    hipEventCreate(&start_ ## name); \
    hipEventCreate(&end_ ## name); \
    float* d_C_INI_ ## name = new float[M * N](); \
    for (int i = 0; i < M; i++) { \
        for (int j = 0; j < N; j++) { \
            d_C_INI_ ## name[i * N + j] = 0; \
        } \
    } \
    for (int i = 0; i < 2; i++) \
    { \
        CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
        name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
    } \
    hipError_t err_t_ ## name = hipGetLastError(); \
    if (err_t_ ## name != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err_t_ ## name) << std::endl; \
    } \
    float milliseconds_ ## name = 0; \
    for (int i = 0; i < NUM_RUNS; i++) \
    { \
        CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
        hipDeviceSynchronize(); \
        hipEventRecord(start_ ## name); \
        name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
        hipEventRecord(end_ ## name); \
        hipEventSynchronize(end_ ## name); \
        float milliseconds_ ## i = 0; \
        hipEventElapsedTime(&milliseconds_ ## i, start_ ## name, end_ ## name); \
        milliseconds_ ## name += milliseconds_ ## i; \
    } \
    hipMemcpy(C, d_C_ ## name, M * N * sizeof(float), hipMemcpyDeviceToHost); \
    std::cout << "Time taken for GEMM (GPU, " << #name <<"): " << milliseconds_ ## name / (float)NUM_RUNS << "ms" << std::endl; \
    hipFree(d_A_ ## name); \
    hipFree(d_B_ ## name); \
    hipFree(d_C_ ## name);

__global__ void gemm_gpu_o0_kernel(float* A, float* B, float *C, int M, int N, int K) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i = 0; i < M; i++) {
            for (int j = 0; j < N; j++) {
                for (int k = 0; k < K; k++) {
                    C[i * N + j]  += A[i * K + k]  * B[k * N + j];
                }
            }
        }
    }
}

void gemm_gpu_o0(float* A, float* B, float* C, int M, int N, int K)
{
    // Init block and grid size
    dim3 blockSize(1);
    dim3 gridSize(1);
    gemm_gpu_o0_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

// The scafolding for optimized GEMM implementations
__global__ void gemm_gpu_o1_kernel(float* A, float* B, float *C, int M, int N, int K) {
    // M x K * K x N = M x N
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((row < M) && (col < N)) {
        for (int e = 0; e < K; e++) {
            Cvalue += A[row * K + e] * B[e * N + col];
        }
        C[row * N + col] = Cvalue;
    }
}
void gemm_gpu_o1(float* A, float* B, float* C, int M, int N, int K)
{
    // Init block and grid size
    dim3 blockSize(16, 16);
    dim3 gridSize(ceil(1.0 * N) / blockSize.x, ceil(1.0 * M) / blockSize.y);
    gemm_gpu_o1_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

#define TILE_WIDTH_O2 16
__global__ void gemm_gpu_o2_kernel(float* A, float* B, float *C, int M, int N, int K) {
    __shared__ float subTileA[TILE_WIDTH_O2][TILE_WIDTH_O2];
    __shared__ float subTileB[TILE_WIDTH_O2][TILE_WIDTH_O2];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by * TILE_WIDTH_O2 + ty;
    int col = bx * TILE_WIDTH_O2 + tx;
    float Pvalue = 0;
    for (int q = 0; q < ceil(1.0 * K / TILE_WIDTH_O2); q++) {
        if ((row < M) && (q * TILE_WIDTH_O2 + tx < K)) 
            subTileA[ty][tx] = A[row * K + q * TILE_WIDTH_O2 + tx];
        else 
            subTileA[ty][tx] = 0;
        if ((col < N) && (q * TILE_WIDTH_O2 + ty < K)) 
            subTileB[ty][tx] = B[(q * TILE_WIDTH_O2 + ty) * N + col];
        else 
            subTileB[ty][tx] = 0;
        __syncthreads();
        for (int k = 0; k < TILE_WIDTH_O2; k++) {
            Pvalue += subTileA[ty][k] * subTileB[k][tx];
        }
        __syncthreads();
    }
    if ((row < M) && (col < N)) 
        C[row * N + col] = Pvalue;
}
void gemm_gpu_o2(float* A, float* B, float* C, int M, int N, int K)
{
    // Init block and grid size
    dim3 blockSize(TILE_WIDTH_O2, TILE_WIDTH_O2);
    dim3 gridSize(ceil((1.0 * N) / TILE_WIDTH_O2), ceil((1.0 * M) / TILE_WIDTH_O2));
    gemm_gpu_o2_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

#define TILE_WIDTH_O3 16
__global__ void gemm_gpu_o3_kernel(float* A, float* B, float *C, int M, int N, int K) {
    __shared__ float subTileA[TILE_WIDTH_O3][TILE_WIDTH_O3];
    __shared__ float subTileB[TILE_WIDTH_O3][TILE_WIDTH_O3];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by * TILE_WIDTH_O3 + ty;
    int col = bx * TILE_WIDTH_O3 + tx;
    float Pvalue = 0;
    for (int q = 0; q < ceil(1.0 * K / TILE_WIDTH_O3); q++) {
        if ((row < M) && (q * TILE_WIDTH_O3 + tx < K)) 
            subTileA[ty][tx] = A[row * K + q * TILE_WIDTH_O3 + tx];
        else 
            subTileA[ty][tx] = 0;
        if ((col < N) && (q * TILE_WIDTH_O3 + ty < K)) 
            subTileB[ty][tx] = B[(q * TILE_WIDTH_O3 + ty) * N + col];
        else 
            subTileB[ty][tx] = 0;
        __syncthreads();
        for (int k = 0; k < TILE_WIDTH_O3; k++) {
            Pvalue += subTileA[ty][k] * subTileB[k][tx];
        }
        __syncthreads();
    }
    if ((row < M) && (col < N)) 
        C[row * N + col] = Pvalue;
}
void gemm_gpu_o3(float* A, float* B, float* C, int M, int N, int K)
{
    // Init block and grid size
    dim3 blockSize(TILE_WIDTH_O3, TILE_WIDTH_O3);
    dim3 gridSize(ceil((1.0 * N) / TILE_WIDTH_O3), ceil((1.0 * M) / TILE_WIDTH_O3));
    gemm_gpu_o3_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

#include "hipblas.h"
void gemm_cublas(float* A, float* B, float* C, int M, int N, int K)
{
    // ref: https://docs.nvidia.com/cuda/cublas/#cublas-t-gemm

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, N, A, K, &beta, C, N);
    hipblasDestroy(handle);
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cout << "Usage: mp1 <M> <N> <K>" << std::endl;
        return 1;
    }

    int M = atoi(argv[1]);
    int N = atoi(argv[2]);
    int K = atoi(argv[3]);

    // int runs = atoi(argv[3]);
    float* A = new float[M * K]();
    float* B = new float[K * N]();
    float* C = new float[M * N]();

    fillRandom(A, M * K);
    fillRandom(B, K * N);

    /// GPU Implementation
        // Check if implementation is correct
    auto ref = Ref();
    float* refC = new float[Ref::M * Ref::N]();
    CHECK(gemm_gpu_o0)
    CHECK(gemm_gpu_o1)
    CHECK(gemm_gpu_o2)
    CHECK(gemm_gpu_o3)
    CHECK(gemm_cublas)

    // Actual run
    TIME(gemm_gpu_o0)
    TIME(gemm_gpu_o1)
    TIME(gemm_gpu_o2)
    TIME(gemm_gpu_o3)
    TIME(gemm_cublas)

    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}